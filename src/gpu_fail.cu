/*
 * @Author: heze
 * @Date: 2021-06-01 00:38:55
 * @LastEditTime: 2021-06-05 00:45:48
 * @Description: 在baseline基础上减了一层循环
 * @FilePath: /src/gpu_fail.cu
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define blockSize 10
#define printArray 0

/**
 * @brief 核函数，
 * 
 * @param width 矩阵列数
 * @param height 矩阵行数
 * @param array 待计算矩阵
 * @param globalResult 存放结果的矩阵
 * @return void
 */
__global__ void cal(int width, int height, int *array, float *globalResult) {
    //索引待计算元素位置
    int ix = threadIdx.x+blockDim.x*blockIdx.x;
    int iy = threadIdx.y+blockDim.y*blockIdx.y;
    int index = ix+iy*width;
    int indexX = index / width;
    int indexY = index % width;
    //计算需计算窗口的四条边分别是哪一行、哪一列，计算窗口总元素个数
    int indexLeft = max(0, indexY-2);
    int indexRight = min(indexY+3, width);
    int indexUp = max(0, indexX-2);
    int indexDown = min(indexX+3, height);
    int indexNum = (indexRight-indexLeft) * (indexDown-indexUp);
    int indexTimes[16]={0};
    float localResult = 0, indexP;

    for(int i=indexUp;i<indexDown;i++){
        for(int j=indexLeft;j<indexRight;j++) {
            indexTimes[array[i * width + j]]++;
        }
    }
    for(int i=0;i<16;i++){
        indexP = (float)indexTimes[i] / indexNum;
        if(indexP!=0.0){
            localResult -= indexP * log2(indexP);
        }
    }
    globalResult[index] = localResult;
}

/**
 * @description: 主函数
 * @param {int} argc  命令行参数个数
 * @param {char const} *argv 命令行参数指针
 * @return {*}
 */
int main(int argc, char const *argv[])
{
    //由运行时的命令行参数获取矩阵的行数和列数，并计算元素个数
    int height=atoi(argv[1]);
    int width=atoi(argv[2]);
    int size=height*width;
    int *host_array,*device_array;
    float *host_result,*device_result;
    //在CPU上分配矩阵和结果的内存
    hipHostMalloc((void **)&host_array,sizeof(int)*size, hipHostMallocDefault);
    hipHostMalloc((void **)&host_result,sizeof(float)*size, hipHostMallocDefault);
    //随机生成矩阵元素
    srand((unsigned)time(0));
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            host_array[i * width + j] = rand()%16;
        }
    }
    if(printArray){
        printf("二维数组：\n");
        for (int i = 0; i < height; i++){
            for (int j = 0; j < width; j++){
                printf("%2d ",host_array[i*width+j]);
            }
            printf("\n");
        } 
    }

    //在GPU上分配矩阵和结果的内存
    hipMalloc((void **) &device_array, sizeof(int)*size);
    hipMalloc((void **) &device_result, sizeof(float)*size);
    hipMemcpy(device_array, host_array, sizeof(int)*size, hipMemcpyHostToDevice);
    
    clock_t start,end;
    //分配线程块大小
    unsigned int grid_rows = (height/blockSize)+1;
    unsigned int grid_cols = (width/blockSize)+1;
    dim3 dimGrid(grid_rows, grid_cols);
    dim3 dimBlock(blockSize, blockSize);
    //调用核函数计算，并在前后计时，最后算出运行时间
    start=clock();
    cal<<<dimGrid, dimBlock>>>(width, height,device_array,device_result);
    hipDeviceSynchronize();
    end=clock();
    double time_gpu=(double)(end-start)/CLOCKS_PER_SEC;
    //将结果从GPU拷贝回CPU，打印信息
    hipMemcpy(host_result,device_result, sizeof(float)*size, hipMemcpyDeviceToHost);
    if(printArray){
        printf("结果：\n");
        for (int i = 0; i < height; i++){
            for (int j = 0; j < width; j++){
                printf("%.5f ",host_result[i*width+j]);
            }
            printf("\n");
        } 
    }
    printf("矩阵维度%dx%d，优化循环在GPU上运行时间: %f ms.\n", height,width,time_gpu*1000);
    hipFree(host_array);
    hipFree(host_result);
    hipFree(device_array);
    hipFree(device_result);
}