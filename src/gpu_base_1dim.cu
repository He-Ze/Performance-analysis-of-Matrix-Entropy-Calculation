/*
 * @Author: heze
 * @Date: 2021-06-01 00:38:55
 * @LastEditTime: 2021-06-05 00:38:49
 * @Description: 使用一维线程的baseline版本
 * @FilePath: /src/gpu_base_1dim.cu
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define printArray 0

/**
 * @brief 核函数，一维线程计算熵值
 * 
 * @param width 矩阵列数
 * @param height 矩阵行数
 * @param array 待计算矩阵
 * @param globalResult 存放结果的矩阵
 * @return void
 */
__global__ void cal(int width, int height, int *array, float *globalResult) {
    //索引待计算元素位置
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int indexX = index / width;
    int indexY = index % width;
    //计算需计算窗口的四条边分别是哪一行、哪一列，计算窗口总元素个数
    int indexLeft = max(0, indexX-2);
    int indexRight = min(indexX+3, width);
    int indexUp = max(0, indexY-2);
    int indexDown = min(indexY+3, height);
    int indexNum = (indexRight-indexLeft) * (indexDown-indexUp);
    int indexTimes;
    float localResult = 0, indexP;

    //每一次循环的任务是计算窗口中有多少等于k的元素，得到结果后计算概率，再取对数相乘后加到结果中
    for(int k=0;k<16;k++) {
        indexTimes = 0;
        for(int i=indexUp;i<indexDown;i++)
            for(int j=indexLeft;j<indexRight;j++) {
                if(array[i * width + j]==k){
                    indexTimes++;
                }
            }
        indexP = (float)indexTimes / indexNum;
        if(indexP!=0){
            localResult -= indexP * log2(indexP);
        }
    }
    globalResult[index] = localResult;
}

/**
 * @description: 主函数
 * @param {int} argc  命令行参数个数
 * @param {char const} *argv 命令行参数指针
 * @return {*}
 */
int main(int argc, char const *argv[])
{
    //由运行时的命令行参数获取矩阵的行数和列数，并计算元素个数
    int height=atoi(argv[1]);
    int width=atoi(argv[2]);
    int size=height*width;
    int *host_array,*device_array;
    float *host_result,*device_result;
    //在CPU上分配矩阵和结果的内存
    hipHostMalloc((void **)&host_array,sizeof(int)*size, hipHostMallocDefault);
    hipHostMalloc((void **)&host_result,sizeof(float)*size, hipHostMallocDefault);
    //随机生成矩阵元素
    srand((unsigned)time(0));
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            host_array[i * width + j] = rand()%16;
        }
    }
    if(printArray){
        printf("二维数组：\n");
        for (int i = 0; i < height; i++){
            for (int j = 0; j < width; j++){
                printf("%2d ",host_array[i*width+j]);
            }
            printf("\n");
        } 
    }
    //在GPU上分配矩阵和结果的内存
    hipMalloc((void **) &device_array, sizeof(int)*size);
    hipMalloc((void **) &device_result, sizeof(float)*size);
    hipMemcpy(device_array, host_array, sizeof(int)*size, hipMemcpyHostToDevice);
    
    clock_t start,end;
    //调用核函数计算，并在前后计时，最后算出运行时间
    start=clock();
    //调用核函数的时候设置一个线程块开1000个线程，块的个数用总元素个数除以1000即可
    cal<<<size/1000, 1000>>>(width, height,device_array,device_result);
    hipDeviceSynchronize();
    end=clock();
    double time_gpu=(double)(end-start)/CLOCKS_PER_SEC;
    //将结果从GPU拷贝回CPU，打印信息
    hipMemcpy(host_result,device_result, sizeof(float)*size, hipMemcpyDeviceToHost);
    if(printArray){
        printf("结果：\n");
        for (int i = 0; i < height; i++){
            for (int j = 0; j < width; j++){
                printf("%.5f ",host_result[i*width+j]);
            }
            printf("\n");
        } 
    }
    printf("矩阵维度%dx%d，一维无优化在GPU上运行时间: %f ms.\n", height,width,time_gpu*1000);
    hipFree(host_array);
    hipFree(host_result);
    hipFree(device_array);
    hipFree(device_result);
}
